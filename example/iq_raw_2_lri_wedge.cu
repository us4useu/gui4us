#include "hip/hip_runtime.h"
#include <cupy/complex.cuh>

#define HIP_PI_F 3.141592654f

__constant__ float zElemConst[256];
__constant__ float xElemConst[256];
__constant__ float tangElemConst[256];


__device__ float xRefract(float const zElem, float const xElem,
                          float const zPix, float const xPix,
                          float const sosInterface, float const sosSample,
                          float const timePrec) {
    float xRefractLo, sinRatioLo, xRefractHi, sinRatioHi, timeOld;
    float xRefractNew, distInterf, distSample, sinRatioNew, timeNew;

    float const cRatio = sosInterface / sosSample;
    // Initial refraction points
    xRefractLo = xElem;
    sinRatioLo = 0.f;

    xRefractHi = xElem - zElem * (xPix - xElem) / (zPix - zElem);
    sinRatioHi = 1.f;

    timeOld = ownHypotf(xRefractHi - xElem, zElem) / sosInterface
            + ownHypotf(xPix - xRefractHi, zPix ) / sosSample;

    // Iterations
    do {
        xRefractNew = xRefractLo + (xRefractHi-xRefractLo)*(cRatio-sinRatioLo)/(sinRatioHi-sinRatioLo);
        distInterf  = ownHypotf(xRefractNew - xElem, zElem);
        distSample  = ownHypotf(xPix - xRefractNew, zPix);
        sinRatioNew = ((xRefractNew - xElem) / distInterf)
                / ((xPix - xRefractNew) / distSample);
        timeNew     = distInterf / sosInterface + distSample / sosSample;
        if (fabs(timeNew-timeOld) < timePrec) {
            break;
        }
        if (sinRatioNew < cRatio) {
            xRefractLo = xRefractNew;
            sinRatioLo = sinRatioNew;
        }
        else {
            xRefractHi = xRefractNew;
            sinRatioHi = sinRatioNew;
        }
        timeOld = timeNew;
    } while(true);
    return xRefractNew;
}


extern "C"
__global__ void
iqRaw2Lri(complex<float> *iqLri, const complex<float> *iqRaw,
          const int nElem,
          const int nSeq, const int nTx, const int nSamp,
          const float *zPix, const int nZPix,
          const float *xPix, const int nXPix,
          float const sos, float const fs, float const fn,
          const float *txApCentZ, const float *txApCentX,
          const int *rxApOrigElem, const int nRx,
          const float minRxTang, const float maxRxTang,
          float const initDel,
          // wedge parameters,
          const float sosInterface, // wedge speed of sound
          const float timePrec
          ) {

    int z = blockIdx.x * blockDim.x + threadIdx.x;
    int x = blockIdx.y * blockDim.y + threadIdx.y;
    int iGlobalTx = blockIdx.z * blockDim.z + threadIdx.z;

    if(z >= nZPix || x >= nXPix || iGlobalTx >= nSeq*nTx) {
        return;
    }
    int iTx = iGlobalTx % nTx;

    int iElem, offset;
    float interpWgh;
    float txTime, rxTime, xRefr, rxTang, txApod, rxApod, time, iSamp;
    float modSin, modCos, pixWgh;
    const float omega = 2 * HIP_PI_F * fn;
    const float sosInv = 1 / sos;
    const float sosInvInterface = 1 / sosInterface;
    const float nSigma = 3; // number of sigmas in half of the apodization Gaussian curve
    const float twoSigSqrInv = nSigma * nSigma * 0.5f;
    const float rngRxTangInv = 2 / (maxRxTang - minRxTang); // inverted half range
    const float centRxTang = (maxRxTang + minRxTang) * 0.5f;
    complex<float> pix(0.0f, 0.0f), samp(0.0f, 0.0f), modFactor;

    int txOffset = iGlobalTx * nSamp * nRx;

    xRefr = xRefract(txApCentZ[iTx], txApCentX[iTx], zPix[z], xPix[x], sosInterface, sos, timePrec);

    txTime = hypotf(0.f - txApCentZ[iTx], xRefr - txApCentX[iTx]) * sosInvInterface
            + hypotf(zPix[z] - 0.f, xPix[x] - xRefr) * sosInv;

    txApod = 1.0f;

    pixWgh = 0.0f;
    pix.real(0.0f);
    pix.imag(0.0f);

    if(txApod != 0.0f) {
        for(int iRx = 0; iRx < nRx; ++iRx) {
            iElem = iRx + rxApOrigElem[iTx];
            if(iElem < 0 || iElem >= nElem) continue;

            xRefr = xRefract(zElemConst[iElem], xElemConst[iElem], zPix[z], xPix[x], sosInterface, sos, timePrec);

            rxTime	= hypotf(xRefr - xElemConst[iElem], 0.f - zElemConst[iElem]) * sosInvInterface
                    + hypotf(xPix[x] - xRefr, zPix[z] - 0.f) * sosInv;


            rxTang = __fdividef(xRefr - xElemConst[iElem], 0.f - zElemConst[iElem]);
            rxTang = __fdividef(rxTang-tangElemConst[iElem], 1.f+rxTang*tangElemConst[iElem]);

            if(rxTang < minRxTang || rxTang > maxRxTang) continue;

            rxApod = (rxTang - centRxTang) * rngRxTangInv;
            rxApod = __expf(-rxApod * rxApod * twoSigSqrInv);

            time = txTime + rxTime + initDel;

            iSamp = time * fs;
            if(iSamp < 0.0f || iSamp >= static_cast<float>(nSamp - 1)) {
                continue;
            }
            offset = txOffset + iRx * nSamp;
            interpWgh = modff(iSamp, &iSamp);
            int intSamp = int(iSamp);

            __sincosf(omega * time, &modSin, &modCos);
            complex<float> modFactor = complex<float>(modCos, modSin);

            samp = iqRaw[offset + intSamp] * (1 - interpWgh) + iqRaw[offset + intSamp + 1] * interpWgh;
            pix += samp * modFactor * rxApod;
            pixWgh += rxApod;
        }
    }
    if(pixWgh == 0.0f) {
        iqLri[z + x*nZPix + iGlobalTx*nZPix*nXPix] = complex<float>(0.0f, 0.0f);
    } else {
        iqLri[z + x*nZPix + iGlobalTx*nZPix*nXPix] = pix / pixWgh * txApod;
    }
}